#include "hip/hip_runtime.h"
#include <kernels.h>
#include <Constants.h>
#include "hip/hip_runtime.h"
#include ""

__global__ void Vector_Addition(const int* dev_a, const int* dev_b, int* dev_c)
{
    //Get the id of thread within a block
    unsigned short tid = blockIdx.x * blockDim.x + threadIdx.x;

        dev_c[tid] = dev_a[tid] + dev_b[tid];
}