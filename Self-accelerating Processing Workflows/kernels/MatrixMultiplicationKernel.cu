#include "hip/hip_runtime.h"
#include <kernels.h>
#include <Constants.h>
#include "hip/hip_runtime.h"
#include ""

__global__ void matrix_multiplication(numericalType1* A, numericalType1* B, numericalType1* C, const int width) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    numericalType1 sum = 0;

    for (int k = 0; k < width; k++) {
        sum += A[y * width + k] * B[k * width + x];
    }

    C[y * width + x] = sum;
}