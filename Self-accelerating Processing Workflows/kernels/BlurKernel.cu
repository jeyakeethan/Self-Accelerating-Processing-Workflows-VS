#include "hip/hip_runtime.h"
#include <kernels.h>
#include <Constants.h>
#include "hip/hip_runtime.h"
#include ""

__global__ void blur_image(unsigned char* input_image, unsigned char* output_image, int width, int height) {

    const unsigned int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int x = offset % width;
    int y = (offset - x) / width;
    int fsize = 5; // Filter size
    if (offset < width * height) {

        float output_red = 0;
        float output_green = 0;
        float output_blue = 0;
        int hits = 0;
        for (int ox = -fsize; ox < fsize + 1; ++ox) {
            for (int oy = -fsize; oy < fsize + 1; ++oy) {
                if ((x + ox) > -1 && (x + ox) < width && (y + oy) > -1 && (y + oy) < height) {
                    const int currentoffset = (offset + ox + oy * width) * 3;
                    output_red += input_image[currentoffset];
                    output_green += input_image[currentoffset + 1];
                    output_blue += input_image[currentoffset + 2];
                    hits++;
                }
            }
        }
        output_image[offset * 3] = output_red / hits;
        output_image[offset * 3 + 1] = output_green / hits;
        output_image[offset * 3 + 2] = output_blue / hits;
    }
}