#include "hip/hip_runtime.h"
#include ""

#include "random_array_generator.cpp"

#include <stdio.h>
#include <iostream>
#include <fstream>

// measure time
#include <windows.h>
#include <time.h>

#include <Constants.h>
#include <ComputationalModel.h>
#include <models/ArrayAddModel.h>
#include <models/ArrayAdd2DModel.h>
#include <random>
#include <string>

using namespace std;
int main()
{	// write logs into file
	//string console_log_name = "../logs/array_addtion_" + CONSOLE_LOG_FILE_NAME;
	//freopen(console_log_name.c_str(), "w", stdout);

	srand(5);		// Random Seed Value

	LARGE_INTEGER start, stop, clockFreq;
	ofstream dataset_file;
	QueryPerformanceFrequency(&clockFreq);
	double delayCPU, delayGPU;
	int elapsedTime;
	int fileNum;
	short favor;
	const int experiment_count = 10;


	/*------- Write Input Nature into File -------*/
	string inputNatureFile = "../ml-datasets/Array-Addition.csv";
	dataset_file.open(inputNatureFile, ios_base::out);

	/*------------- Single dimension vector addition ------------*/
	cout << "One Dimension experiments started" << endl;

	ArrayAdditionModel<numericalType1> arrayAdditionModel(6);

	const int number_entries = 1000;
	numericalType1* arraySet1 [experiment_count];
	numericalType1* arraySet2 [experiment_count];
	numericalType1* outputs [experiment_count];
	int arrayLength[number_entries];
	int x, y, z, k, i, length;
	int step = 10000;
	for (i = 1; i < number_entries; i++) {
		length = step * i;
		arrayLength[i] = length;
		for (x = 0; x < experiment_count; x++) {
			arraySet1[x] = generate_1d_array(length);
			arraySet2[x] = generate_1d_array(length);
			outputs[x] = new numericalType1[length];
		}

		/*-------- CPU Time - ArrayAdditionModel --------*/
		QueryPerformanceCounter(&start);
		for (x = 0; x < experiment_count; x++) {
			arrayAdditionModel.SetData(arraySet1[x], arraySet2[x], outputs[x], length);
			arrayAdditionModel.execute(1);
		}
		QueryPerformanceCounter(&stop);
		delayCPU = (double)(stop.QuadPart - start.QuadPart);
		cout << "CPU Time: " << delayCPU << ", ";

		/*-------- GPU Time - ArrayAdditionModel --------*/
		QueryPerformanceCounter(&start);
		for (x = 0; x < experiment_count; x++) {
			arrayAdditionModel.SetData(arraySet1[x], arraySet2[x], outputs[x], length);
			arrayAdditionModel.execute(2);
		}
		QueryPerformanceCounter(&stop);
		delayGPU = (double)(stop.QuadPart - start.QuadPart);
		cout << "GPU Time: " << delayGPU << ", " << endl;

		dataset_file << length << "," << (delayGPU > delayCPU ? 0 : 1) << endl;

		/*************Free Host Memory**************/
		for (x = 0; x < experiment_count; x++) {
			delete[] arraySet1[x];
			delete[] arraySet2[x];
			delete[] outputs[x];
		}
	}
	dataset_file.close();
	return 0;
}
