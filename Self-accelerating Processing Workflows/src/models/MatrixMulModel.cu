#include "hip/hip_runtime.h"
#include <models/MatrixMulModel.h>
#include <kernels.h>
#include <omp.h>

#include "hip/hip_runtime.h"
#include ""

#include <Constants.h>
#include <iostream>

#ifndef _MATRIXMULTIPLICATIONMODEL_CPP_
#define _MATRIXMULTIPLICATIONMODEL_CPP_

using namespace std;

template <class T>
MatrixMultiplicationModel<T>::MatrixMultiplicationModel(int CPUCores):ComputationalModel(CPUCores, "matrix-multiplication") { }

template <class T>
MatrixMultiplicationModel<T>::~MatrixMultiplicationModel() {}


template <class T>
void MatrixMultiplicationModel<T>::CPUImplementation() {
	// log mode to see the flow of execution
	CPUGPULOG << 0;

	int x = localMD->x, y = localMD->y, z = localMD->z;
	
	//implement using multi threads
#pragma omp parallel num_threads(CPUCores)
		{
#pragma omp for
			for (int i = 0; i < x; i++) {
				for (int j = 0; j < z; j++) {
					T sum = 0;
					for (int k = 0; k < y; k++) {
						sum += localA[y * i + k] * localB[j + z * k];
					}
					localC[z * i + j] = sum;
				}
			}
		}
#pragma omp barrier
}

template <class T>
void MatrixMultiplicationModel<T>::GPUImplementation() {
	// log mode to see the flow of execution
	CPUGPULOG << 1;

	//Device array
	numericalType1 *dev_a, *dev_b, *dev_c;

	int l1 = localMD->x * localMD->y * sizeof(numericalType1);
	int l2 = localMD->y * localMD->z * sizeof(numericalType1);
	int l3 = localMD->x * localMD->z * sizeof(numericalType1);

	//Allocate the memory on the GPU
	hipMalloc((void**)&dev_a, l1);
	hipMalloc((void**)&dev_b, l2);
	hipMalloc((void**)&dev_c, l3);

	//Copy Host array to Device array
	hipMemcpy(dev_a, localA, l1, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, localB, l2, hipMemcpyHostToDevice);
	// Execute the kernel
	// define grid and thread block sizes

	dim3 dimGrid(32, 1024), dimBlock(32);
	matrix_multiplication << < dimGrid, dimBlock >> > (dev_a, dev_b, dev_c, localMD->y, localMD->z);

	//Copy back to Host array from Device array
	hipMemcpy(localC, dev_c, l3, hipMemcpyDeviceToHost);

	//Free the Device array memory
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	//sychronize to confirm that results have been computed and copied back
	hipDeviceSynchronize();
}


// retrive attributes
template <class T>
vector<float> MatrixMultiplicationModel<T>::getAttributes() {
	return *attr;
}

template <class T>
vector<float> MatrixMultiplicationModel<T>::getAttributesBatch() {
	return *attr;
}
#endif // _MATRIXMULTIPLICATIONMODEL_CPP_