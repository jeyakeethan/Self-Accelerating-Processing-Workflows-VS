#include <models/ArrayAdditionModel.h>
#include <kernels.h>

#include "hip/hip_runtime.h"
#include ""

#include <Constants.h>
#include <iostream>

#ifndef _ARRAYADDITIONMODEL_CPP_
#define _ARRAYADDITIONMODEL_CPP_

using namespace std;

template <class T>
ArrayAdditionModel<T>::ArrayAdditionModel(int CPUCores): ComputationalModel(CPUCores) {
}

template <class T>
ArrayAdditionModel<T>::~ArrayAdditionModel() {}

template <class T>
void ArrayAdditionModel<T>::CPUImplementation(){
    for(int x = 0; x < localL; x++){
        //cout << localA[x] << "," << localB[x] << ",";
        localC[x] = localA[x] + localB[x];
    }
}

template <class T>
void ArrayAdditionModel<T>::GPUImplementation(){
    //Device array
    int *dev_a , *dev_b, *dev_c;
    //Allocate the memory on the GPU
    hipMalloc((void **)&dev_a , localL *sizeof(int));
    hipMalloc((void **)&dev_b , localL *sizeof(int));
    hipMalloc((void **)&dev_c , localL *sizeof(int));
    //Copy Host array to Device array
    hipMemcpy (dev_a , localA , localL *sizeof(int) , hipMemcpyHostToDevice);
    hipMemcpy (dev_b , localB , localL *sizeof(int) , hipMemcpyHostToDevice);
    // Execute the kernel

    Vector_Addition <<< localL / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>> (dev_a, dev_b, dev_c);
    //Copy back to Host array from Device array
    hipMemcpy(localC , dev_c , localL *sizeof(int) , hipMemcpyDeviceToHost);
    //Free the Device array memory
    hipFree (dev_a);
    hipFree (dev_b);
    hipFree (dev_c);
}

#endif // ARRAYADDITIONMODEL_CPP