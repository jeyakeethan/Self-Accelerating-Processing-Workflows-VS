#include "hip/hip_runtime.h"
#include <models/ArrayAddModel.h>
#include <kernels.h>
#include <omp.h>

#include "hip/hip_runtime.h"
#include ""

#include <Constants.h>
#include <iostream>

#ifndef _ARRAYADDMODEL_CPP_
#define _ARRAYADDMODEL_CPP_

using namespace std;

template <class T>
ArrayAdditionModel<T>::ArrayAdditionModel(int CPUCores): ComputationalModel(CPUCores, "Array-Addition") {
}

template <class T>
ArrayAdditionModel<T>::~ArrayAdditionModel() {}

template <class T>
void ArrayAdditionModel<T>::CPUImplementation(){
#pragma omp parallel num_threads(CPUCores)
    {
#pragma omp for
    for(int x = 0; x < localL; x++){
        //cout << localA[x] << "," << localB[x] << ",";
        localC[x] = localA[x] + localB[x];
    }
#pragma omp barrier
    }
}

template <class T>
void ArrayAdditionModel<T>::GPUImplementation(){
    //Device array
    int *dev_a , *dev_b, *dev_c;
    //Allocate the memory on the GPU
    hipMalloc((void **)&dev_a , localL *sizeof(int));
    hipMalloc((void **)&dev_b , localL *sizeof(int));
    hipMalloc((void **)&dev_c , localL *sizeof(int));
    //Copy Host array to Device array
    hipMemcpy (dev_a , localA , localL *sizeof(int) , hipMemcpyHostToDevice);
    hipMemcpy (dev_b , localB , localL *sizeof(int) , hipMemcpyHostToDevice);
    // Execute the kernel

    dim3 blockDims(THREADS_PER_BLOCK, 1, 1);
    dim3 gridDims((unsigned int)ceil((double)(localL / blockDims.x)), 1, 1);
    Vector_Addition << < blockDims, gridDims >> > (dev_a, dev_b, dev_c);
    //Copy back to Host array from Device array
    hipMemcpy(localC , dev_c , localL *sizeof(int) , hipMemcpyDeviceToHost);
    //Free the Device array memory
    hipFree (dev_a);
    hipFree (dev_b);
    hipFree (dev_c);
}

template <class T>
vector<float>* ArrayAdditionModel<T>::getAttributes(){
    return new vector<float>{ 1, float(localL) };
}

#endif //ARRAYADDMODEL_CPP