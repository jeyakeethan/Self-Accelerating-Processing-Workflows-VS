#include "hip/hip_runtime.h"
#include <models/ComplexModel.h>
#include <kernels.h>
#include <omp.h>

#include "hip/hip_runtime.h"
#include ""

#include <Constants.h>
#include <iostream>

#ifndef _COMPLEX_MODEL_CPP_
#define _COMPLEX_MODEL_CPP_

using namespace std;

template <class T>
ComplexModel<T>::ComplexModel(int CPUCores) :ComputationalModel(CPUCores, "matrix-multiplication") {
	//super(CPUCores);
}

template <class T>
ComplexModel<T>::~ComplexModel() {}


template <class T>
void ComplexModel<T>::CPUImplementation() {
	// log mode to see the flow of execution
	CPUGPULOG << 0;
	int x = localMD->x, y = localMD->y, z = localMD->z;

	//implement using multi threads
#pragma omp parallel num_threads(CPUCores)
	{
#pragma omp for
		for (int i = 0; i < x; i++) {
			for (int j = 0; j < z; j++) {
				T sum = 0;
				for (int k = 0; k < y; k++) {
					sum += localA[y * i + k] * localB[j + z * k];
				}
				int index = z * i + j;
				localC[index] = sum + localX[index];
			}
		}
	}
#pragma omp barrier
}

template <class T>
void ComplexModel<T>::GPUImplementation() {
	// log mode to see the flow of execution
	CPUGPULOG << 1;

	//Device array
	numericalType1* dev_a, * dev_b, * dev_out, * dev_x;
	int x = localMD->x, y = localMD->y, z = localMD->z;
	int l1 = x * y * sizeof(numericalType1);
	int l2 = y * z * sizeof(numericalType1);
	int l3 = x * z * sizeof(numericalType1);

	//Allocate the memory on the GPU
	hipMalloc((void**)&dev_a, l1);
	hipMalloc((void**)&dev_b, l2);
	hipMalloc((void**)&dev_out, l3);
	hipMalloc((void**)&dev_x, l3);

	//Copy Host array to Device array
	hipMemcpy(dev_a, localA, l1, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, localB, l2, hipMemcpyHostToDevice);
	hipMemcpy(dev_x, localB, l3, hipMemcpyHostToDevice);
	// Execute the kernel
	// define grid and thread block sizes

	dim3 dimGrid(32, 1024), dimBlock(32);
	complex_model << < dimGrid, dimBlock >> > (dev_a, dev_b, dev_x, dev_out, y, z);

	//Copy back to Host array from Device array
	hipMemcpy(localC, dev_out, l3, hipMemcpyDeviceToHost);

	//Free the Device array memory
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_x);
	hipFree(dev_out);

	//sychronize to confirm that results have been computed and copied back
	hipDeviceSynchronize();
}


// retrive attributes
template <class T>
vector<float>* ComplexModel<T>::getAttributes() {
	return attr;
}

template <class T>
vector<float>* ComplexModel<T>::getAttributesBatch() {
	return attr;
}
#endif // _COMPLEX_MODEL_CPP_