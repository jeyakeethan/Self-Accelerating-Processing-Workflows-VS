#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "random_array_generator.cpp"

#include "pandas.h"
#include <stdio.h>
#include <iostream>
#include <fstream>

// measure time
#include <windows.h>
#include <time.h>

#include <Constants.h>
#include <ComputationalModel.h>
#include <models/ArrayAddModel.h>
#include <models/ArrayAdd2DModel.h>
#include <random>
#include <string>

using namespace std;
int main()
{
	//string console_log_name = "../logs/Array_addtion_" + CONSOLE_LOG_FILE_NAME;
	//freopen(console_log_name.c_str(), "w", stdout);	// write logs into file

	srand(5);		// Random Seed Value

	LARGE_INTEGER start, stop, clockFreq;
	ofstream input_nature_file;
	ofstream time_log_file;
	QueryPerformanceFrequency(&clockFreq);
	double delay;
	int elapsedTime;
	int fileNum;
	short favor;


	/*------- Write Input Nature into File -------*/
	string inputNatureFile = "../logs/Array_addition_Input Nature.csv"; fileNum = 0;
	while (FILE* file = fopen(inputNatureFile.c_str(), "r")) {
		fclose(file);
		inputNatureFile = "../logs/Array_addition_Input Nature_" + to_string(++fileNum) + ".csv";
	}
	input_nature_file.open(inputNatureFile, ios_base::out);

	string timeLogFile = "../logs/Array_addition_Time.txt"; fileNum = 0;
	while (FILE* file = fopen(timeLogFile.c_str(), "r")) {
		fclose(file);
		timeLogFile = "../logs/Array_addition_Time_" + to_string(++fileNum) + ".txt";
	}
	time_log_file.open(timeLogFile);

/*------------- Single dimension vector addition ------------*/
cout << "One Dimension experiments started" << endl;
input_nature_file << "One Dimension experiments started" << endl;
time_log_file << "One Dimension experiments started" << endl;

	ArrayAdditionModel<numericalType1> arrayAdditionModel(6);

	const int BOUNDARY_POINT = 100000;
	numericalType1* arraySet1[EXPERIMENT_COUNT];
	numericalType1* arraySet2[EXPERIMENT_COUNT];
	numericalType1* outputs[EXPERIMENT_COUNT];
	int arrayLength[EXPERIMENT_COUNT];
	int x, y, z, k, length;

	for (x = 0; x < EXPERIMENT_COUNT; x++) {
		favor = rand() % 2;
		if (favor == 0) length = rand() % BOUNDARY_POINT + 1;
		else length = rand() % BOUNDARY_POINT + BOUNDARY_POINT + 1;
		arrayLength[x] = length;
		arraySet1[x] = generate_1d_array(length);
		arraySet2[x] = generate_1d_array(length);
		outputs[x] = new numericalType1[length];

		input_nature_file << length << ",";		// log input nature
	}

	// -------- Framework - ArrayAdditionModel --------
	QueryPerformanceCounter(&start);
	for (x = 0; x < EXPERIMENT_COUNT; x++) {
		int len = arrayLength[x];
		arrayAdditionModel.invoke(arraySet1[x], arraySet2[x], outputs[x], len);
		arrayAdditionModel.execute();
	}
	QueryPerformanceCounter(&stop);
	delay = (double)(stop.QuadPart - start.QuadPart) / (double)clockFreq.QuadPart;
	elapsedTime = int(delay * 1000);
	cout << "\nAuto Time: " << elapsedTime << " ms" << endl << endl;
	time_log_file << "Auto Time: " << elapsedTime << " ms" << endl << endl;

	// -------- CPU Time - ArrayAdditionModel --------
	QueryPerformanceCounter(&start);
	for (x = 0; x < EXPERIMENT_COUNT; x++) {
		int len = arrayLength[x];
		arrayAdditionModel.invoke(arraySet1[x], arraySet2[x], outputs[x], len);
		arrayAdditionModel.execute(1);
	}
	QueryPerformanceCounter(&stop);
	delay = (double)(stop.QuadPart - start.QuadPart) / (double)clockFreq.QuadPart;
	elapsedTime = int(delay * 1000);
	cout << "CPU Time: " << elapsedTime << " ms" << endl << endl;
	time_log_file << "CPU Time: " << elapsedTime << " ms" << endl << endl;

	// -------- GPU Time - ArrayAdditionModel --------
	QueryPerformanceCounter(&start);
	for (x = 0; x < EXPERIMENT_COUNT; x++) {
		int len = arrayLength[x];
		arrayAdditionModel.invoke(arraySet1[x], arraySet2[x], outputs[x], len);
		arrayAdditionModel.execute(2);
	}
	QueryPerformanceCounter(&stop);
	delay = (double)(stop.QuadPart - start.QuadPart) / (double)clockFreq.QuadPart;
	elapsedTime = int(delay * 1000);
	cout << "GPU Time: " << elapsedTime << " ms" << endl << endl;
	time_log_file << "GPU Time: " << elapsedTime << " ms" << endl << endl;

	// *************Free Host Memory**************
	for (x = 0; x < EXPERIMENT_COUNT; x++) {
		delete[] arraySet1[x];
		delete[] arraySet2[x];
		delete[] outputs[x];
	}

/*------------- Two dimension vector addition ------------*/
cout << "Two Dimension experiments started" << endl;
input_nature_file << "Two Dimension experiments started" << endl;
time_log_file << "Two Dimension experiments started" << endl;

	ArrayAddition2DModel<numericalType1> arrayAddition2DModel(6);

	numericalType1* arraySetB1[EXPERIMENT_COUNT];
	numericalType1* arraySetB2[EXPERIMENT_COUNT];

	// load related dimesion spaces
	const int dim_space_len_2d = 10;
	
	int index_g;

	myDim2 cpu_dim_space_2d[dim_space_len_2d];
	myDim2 gpu_dim_space_2d[dim_space_len_2d];
	pandas::Dataset dataset = pandas::ReadCSV("../ml-datasets/Array-Addition2D.csv", ',', -1, 1000);
	for (x = 0; x < dim_space_len_2d; x++) {
		len_dataset = dataset.labels.size();

		cpu_dim_space_2d[x].x = dataset.features.at(x).at(0);
		cpu_dim_space_2d[x].y = dataset.features.at(x).at(1);

		index_g = len_dataset - dim_space_len_2d + len_dataset
		gpu_dim_space_2d[x].x = dataset.features.at(index_g).at(0);
		gpu_dim_space_2d[x].y = dataset.features.at(index_g).at(1);
	}
	myDim2 dimensions[EXPERIMENT_COUNT];
	numericalType1** outputB;
	int dim_index;
	myDim2 dimension;

	for (x = 0; x < EXPERIMENT_COUNT; x++) {
		favor = rand() % 2;
		dim_index = rand() % dim_space_len_2d;
		if (favor == 0) dimension = cpu_dim_space_2d[dim_index];
		else dimension = gpu_dim_space_2d[dim_index];

		dimensions[x] = dimension;
		length = dimension.x * dimension.y;
		arraySetB1[x] = generate_1d_array(length);
		arraySetB2[x] = generate_1d_array(length);
		outputB[x] = new numericalType1[length];

		input_nature_file << "[" << dimension.x << "," << dimension.y << "]" << ", " << endl;		// log input nature
	}

	// -------- Framework - ArrayAdditionModel --------
	QueryPerformanceCounter(&start);
	for (x = 0; x < EXPERIMENT_COUNT; x++) {
		arrayAddition2DModel.invoke(arraySetB1[x], arraySetB2[x], outputB[x], dimensions[x].x, dimensions[x].y);
		arrayAddition2DModel.execute();
	}
	QueryPerformanceCounter(&stop);
	delay = (double)(stop.QuadPart - start.QuadPart) / (double)clockFreq.QuadPart;
	elapsedTime = int(delay * 1000);
	cout << "\nAuto Time: " << elapsedTime << " ms" << endl << endl;
	time_log_file << "Auto Time: " << elapsedTime << " ms" << endl << endl;

	// -------- CPU Time - ArrayAdditionModel --------
	QueryPerformanceCounter(&start);
	for (x = 0; x < EXPERIMENT_COUNT; x++) {
		arrayAddition2DModel.invoke(arraySetB1[x], arraySetB2[x], outputB[x], dimensions[x].x, dimensions[x].y);
		arrayAddition2DModel.execute(1);
	}
	QueryPerformanceCounter(&stop);
	delay = (double)(stop.QuadPart - start.QuadPart) / (double)clockFreq.QuadPart;
	elapsedTime = int(delay * 1000);
	cout << "CPU Time: " << elapsedTime << " ms" << endl << endl;
	time_log_file << "CPU Time: " << elapsedTime << " ms" << endl << endl;

	// -------- GPU Time - ArrayAdditionModel --------
	QueryPerformanceCounter(&start);
	for (x = 0; x < EXPERIMENT_COUNT; x++) {
		arrayAddition2DModel.invoke(arraySetB1[x], arraySetB2[x], outputB[x], dimensions[x].x, dimensions[x].y);
		arrayAddition2DModel.execute(2);
	}
	QueryPerformanceCounter(&stop);
	delay = (double)(stop.QuadPart - start.QuadPart) / (double)clockFreq.QuadPart;
	elapsedTime = int(delay * 1000);
	cout << "GPU Time: " << elapsedTime << " ms" << endl << endl;
	time_log_file << "GPU Time: " << elapsedTime << " ms" << endl << endl;

	// ************Free Host Memory**************
	for (x = 0; x < EXPERIMENT_COUNT; x++) {
		delete[] arraySetB1[x];
		delete[] arraySetB2[x];
		delete[] outputB[x];
	}
	delete[] arraySetB1;
	delete[] arraySetB2;
	delete[] outputB;
	delete[] dimensions;
	delete[] cpu_dim_space_2d;
	delete[] gpu_dim_space_2d;


/*------------- THree dimension vector addition ------------
cout << "Three Dimension experiments started" << endl;
input_nature_file << "Three Dimension experiments started" << endl;
time_log_file << "Three Dimension experiments started" << endl;

	ArrayAddition3DModel<numericalType1> arrayAddition3DModel(6);

	numericalType1* arraySetC1[EXPERIMENT_COUNT];
	numericalType1* arraySetC2 [EXPERIMENT_COUNT];

	// load related dimesion spaces
	const int dim_space_len_3d = 10;

	myDim3 cpu_dim_space_3d[dim_space_len_3d];
	myDim3 gpu_dim_space_3d[dim_space_len_3d];
	//TO DO

	myDim3 dimensions_3d[EXPERIMENT_COUNT];
	myDim3 dimension_3d;


	numericalType1** outputB;

	for (x = 0; x < EXPERIMENT_COUNT; x++) {
		favor = rand() % 2;
		dim_index = rand() % dim_space_len_3d;
		if (favor == 0) dimension_3d = cpu_dim_space_3d[dim_index];
		else dimension_3d = gpu_dim_space_3d[dim_index];
		dimensions_3d[x] = dimension_3d;

		length = dimension_3d.x * dimension_3d.y * dimension_3d.z;
		arraySetC1[x] = generate_1d_array(length);
		arraySetC2[x] = generate_1d_array(length);

		input_nature_file << "[" << dimension_3d.x << "," << dimension_3d.y <<  "," << dimension_3d.z <<"]" << ", " << endl;		// log input nature
	}

	numericalType1**** outputC;
	for (x = 0; x < EXPERIMENT_COUNT; x++) {
		myDim3 dim_3d = dimensions_3d[x];
		outputC[x] = new numericalType1 **[dim_3d.x];
		for (y = 0; y < dim_3d.x; y++) {
			outputC[x][y] = new numericalType1 * [dim_3d.y];
			for (z = 0; z < dim_3d.y; z++)
				outputC[x][y][z] = new numericalType1[dim_3d.z];
		}
	}
	//-------- Framework - ArrayAdditionModel --------
	QueryPerformanceCounter(&start);
	for (x = 0; x < EXPERIMENT_COUNT; x++) {
		arrayAddition3DModel.invoke(arraySetC1[x], arraySetC2[x], outputC[x], dimensions_3d[x]);
		arrayAddition3DModel.execute();
	}
	QueryPerformanceCounter(&stop);
	delay = (double)(stop.QuadPart - start.QuadPart) / (double)clockFreq.QuadPart;
	elapsedTime = int(delay * 1000);
	cout << "\nAuto Time: " << elapsedTime << " ms" << endl << endl;
	time_log_file << "Auto Time: " << elapsedTime << " ms" << endl << endl;

	//-------- CPU Time - ArrayAdditionModel --------
	QueryPerformanceCounter(&start);
	for (x = 0; x < EXPERIMENT_COUNT; x++) {
		arrayAddition3DModel.invoke(arraySetC1[x], arraySetC2[x], outputC[x], dimensions_3d[x]);
		arrayAddition3DModel.execute(1);
	}
	QueryPerformanceCounter(&stop);
	delay = (double)(stop.QuadPart - start.QuadPart) / (double)clockFreq.QuadPart;
	elapsedTime = int(delay * 1000);
	cout << "CPU Time: " << elapsedTime << " ms" << endl << endl;
	time_log_file << "CPU Time: " << elapsedTime << " ms" << endl << endl;

	//-------- GPU Time - ArrayAdditionModel --------
	QueryPerformanceCounter(&start);
	for (x = 0; x < EXPERIMENT_COUNT; x++) {
		arrayAddition3DModel.invoke(arraySetC1[x], arraySetC2[x], outputC[x], dimensions_3d[x]);
		arrayAddition3DModel.execute(2);
	}
	QueryPerformanceCounter(&stop);
	delay = (double)(stop.QuadPart - start.QuadPart) / (double)clockFreq.QuadPart;
	elapsedTime = int(delay * 1000);
	cout << "GPU Time: " << elapsedTime << " ms" << endl << endl;
	time_log_file << "GPU Time: " << elapsedTime << " ms" << endl << endl;

	//*************Free Host Memory**************
	for (x = 0; x < EXPERIMENT_COUNT; x++) {
		delete[] arraySetC1[x];
		delete[] arraySetC2[x];
		delete[] outputC[x];
	}
	delete[] arraySetC1;
	delete[] arraySetC2;
	delete[] outputC;
	delete[] dimensions_3d;
	delete[] cpu_dim_space_3d;
	delete[] gpu_dim_space_3d;
*/



/*------------- Size based experiments started ------------*/
cout << "Size based experiments started" << endl;
input_nature_file << "Size based experiments started" << endl;
time_log_file << "Size based experiments started" << endl;

	const int start_len = 10000, len_step = 10000;	// BOUNDARY_POINT = 100000; above
	for (length = 0; length < BOUNDARY_POINT * 2; length += len_step) {
		input_nature_file << length << endl;		// log input nature

		for (x = 0; x < EXPERIMENT_COUNT; x++) {
			arraySet1[x] = generate_1d_array(length);
			arraySet2[x] = generate_1d_array(length);
			outputs[x] = new numericalType1(length);
		}

		// -------- Framework - ArrayAdditionModel --------
		QueryPerformanceCounter(&start);
		for (x = 0; x < EXPERIMENT_COUNT; x++) {
			arrayAdditionModel.invoke(arraySet1[x], arraySet2[x], outputs[x], length);
			arrayAdditionModel.execute();
		}
		QueryPerformanceCounter(&stop);
		delay = (double)(stop.QuadPart - start.QuadPart) / (double)clockFreq.QuadPart;
		elapsedTime = int(delay * 1000);
		cout << "\nAuto Time: " << elapsedTime << " ms" << endl << endl;
		time_log_file << "Auto Time: " << elapsedTime << " ms" << endl << endl;

		// -------- CPU Time - ArrayAdditionModel --------
		QueryPerformanceCounter(&start);
		for (x = 0; x < EXPERIMENT_COUNT; x++) {
			arrayAdditionModel.invoke(arraySet1[x], arraySet2[x], outputs[x], length);
			arrayAdditionModel.execute(1);
		}
		QueryPerformanceCounter(&stop);
		delay = (double)(stop.QuadPart - start.QuadPart) / (double)clockFreq.QuadPart;
		elapsedTime = int(delay * 1000);
		cout << "CPU Time: " << elapsedTime << " ms" << endl << endl;
		time_log_file << "CPU Time: " << elapsedTime << " ms" << endl << endl;

		// -------- GPU Time - ArrayAdditionModel --------
		QueryPerformanceCounter(&start);
		for (x = 0; x < EXPERIMENT_COUNT; x++) {
			arrayAdditionModel.invoke(arraySet1[x], arraySet2[x], outputs[x], length);
			arrayAdditionModel.execute(2);
		}
		QueryPerformanceCounter(&stop);
		delay = (double)(stop.QuadPart - start.QuadPart) / (double)clockFreq.QuadPart;
		elapsedTime = int(delay * 1000);
		cout << "GPU Time: " << elapsedTime << " ms" << endl << endl;
		time_log_file << "GPU Time: " << elapsedTime << " ms" << endl << endl;

		//************Free Host Memory**************
		for (x = 0; x < EXPERIMENT_COUNT; x++) {
			delete[] arraySet1[x];
			delete[] arraySet2[x];
			delete[] outputs[x];
		}
	}


	input_nature_file.close();
	time_log_file.close();


	return 0;
}
