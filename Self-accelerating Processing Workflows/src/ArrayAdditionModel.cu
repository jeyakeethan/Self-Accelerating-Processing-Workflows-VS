#include <Constants.h>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

#include <ArrayAdditionModel.h>
#include <kernels.h>

ArrayAdditionModel::ArrayAdditionModel() {}
ArrayAdditionModel::ArrayAdditionModel(int *in1, int *in2, int *out, int length): localA(in1), localB(in2), localC(out), localL(length) { }
ArrayAdditionModel::~ArrayAdditionModel() {}

void ArrayAdditionModel::CPUImplementation(){
    printf("Hello CPU IMPL \n");
    for(int x = 0; x < localL; x++){
        localC[x] = localA[x] + localB[x];
    }
}
void ArrayAdditionModel::GPUImplementation(){
    printf("Hello GPU IMPL \n");
    //Device array
    int *dev_a , *dev_b, *dev_c;
    //Allocate the memory on the GPU
    hipMalloc((void **)&dev_a , N*sizeof(int));
    hipMalloc((void **)&dev_b , N*sizeof(int));
    hipMalloc((void **)&dev_c , N*sizeof(int));
    //Copy Host array to Device array
    hipMemcpy (dev_a , localA , N*sizeof(int) , hipMemcpyHostToDevice);
    hipMemcpy (dev_b , localB , N*sizeof(int) , hipMemcpyHostToDevice);
    // Execute the kernel

    Vector_Addition <<< N / THREATS_PER_BLOCK, THREATS_PER_BLOCK >>> (dev_a, dev_b, dev_c);
    //Copy back to Host array from Device array
    hipMemcpy(localC , dev_c , N*sizeof(int) , hipMemcpyDeviceToHost);
    //Free the Device array memory
    hipFree (dev_a);
    hipFree (dev_b);
    hipFree (dev_c);
}
