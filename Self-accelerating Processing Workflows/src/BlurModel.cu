#include "hip/hip_runtime.h"
#include <BlurModel.h>
#include <kernels.h>
#include <omp.h>

#include "hip/hip_runtime.h"
#include ""

#include <Constants.h>
#include <iostream>

#ifndef _BLURMODEL_CPP_
#define _BLURMODEL_CPP_

using namespace std;

template <class T>
BlurModel<T>::BlurModel(int CPUCores):ComputationalModel(CPUCores) {
	//super(CPUCores);
}

template <class T>
BlurModel<T>::~BlurModel() {}


// retrive influenced attributes
template <class T>
vector<float>* BlurModel<T>::getAttributes() {
	return attr;
}

template <class T>
void BlurModel<T>::CPUImplementation() {
	// log mode to see the flow of execution
	CPUGPULOG << 0;

	/*	
	//implement using multi threads
#pragma omp parallel num_threads(CPUCores)
		{
#pragma omp for
			for (int i = 0; i < localMD->x; i++) {
				for (int j = 0; j < localMD->z; j++) {
					T sum = 0;
					for (int k = 0; k < localMD->y; k++) {
						sum += localA[localMD->y * i + k] * localB[j + localMD->z * k];
					}
					localC[localMD->z * i + j] = sum;
				}
			}
#pragma omp barrier
		}
		*/
}

template <class T>
void BlurModel<T>::GPUImplementation() {
	// log mode to see the flow of execution
	CPUGPULOG << 1;

	unsigned char* dev_input;
	unsigned char* dev_output;
	int size = width * height * 3;
	getError(hipMalloc((void**)&dev_input, size * sizeof(unsigned char)));
	getError(hipMemcpy(dev_input, input_image, size * sizeof(unsigned char), hipMemcpyHostToDevice));

	getError(hipMalloc((void**)&dev_output, size * sizeof(unsigned char)));

	dim3 blockDims(512, 1, 1);
	dim3 gridDims((unsigned int)ceil((double)(size / blockDims.x)), 1, 1);

	blur_image << <gridDims, blockDims >> > (dev_input, dev_output, width, height);


	getError(hipMemcpy(output_image, dev_output, size * sizeof(unsigned char), hipMemcpyDeviceToHost));

	getError(hipFree(dev_input));
	getError(hipFree(dev_output));
}

template <class T>
void BlurModel<T>::getError(hipError_t err) {
    if (err != hipSuccess) {
        cout << "Error " << hipGetErrorString(err) << endl;
    }
}
#endif // _BLURMODEL_CPP_