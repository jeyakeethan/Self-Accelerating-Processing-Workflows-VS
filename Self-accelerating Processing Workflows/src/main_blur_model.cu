``#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <fstream>

#include <Constants.h>
#include <models/BlurModel.h>

#include <stdio.h>

#include "lodepng.h"

using namespace std;
int main() {
    
    const char* input_file = "../source/Lenna.png";
    const char* output_file = "../output/output.png";

    vector<unsigned char> in_image;
    unsigned int width, height;

    // Load the data
    unsigned error = lodepng::decode(in_image, width, height, input_file);
    if (error) cout << "decoder error " << error << ": " << lodepng_error_text(error) << endl;

    // Prepare the data
    unsigned char *input_image = new unsigned char[(in_image.size() * 3) / 4];
    unsigned char *output_image = new unsigned char[(in_image.size() * 3) / 4];
    int where = 0;
    for (int i = 0; i < in_image.size(); ++i) {
        if ((i + 1) % 4 != 0) {
            input_image[where] = in_image.at(i);
            output_image[where] = 255;
            where++;
        }
    }

    // Run the filter on it
    BlurModel<unsigned char> blurModel(6);
    blurModel.invoke(input_image, output_image, 512, 512);
    blurModel.execute(1);

    // Prepare data for output
    vector<unsigned char> out_image;
    for (int i = 0; i < (in_image.size() * 3) / 4; ++i) {
        out_image.push_back(output_image[i]);
        if ((i + 1) % 3 == 0) {
            out_image.push_back(255);
        }
    }

    // Output the data
    error = lodepng::encode(output_file, out_image, width, height);

    //if there's an error, display it
    if (error) cout << "encoder error " << error << ": " << lodepng_error_text(error) << endl;

    delete[] input_image;
    delete[] output_image;
    return 0;

}